#include "hip/hip_runtime.h"
#include <cuda/pipeline>
#include <hip/hip_cooperative_groups.h>

#include <iostream>
#include <array>
#include <vector>
#include "structs_n_stuff.cuh"

namespace cg = cooperative_groups;

#define CUDA_CHECK(status)                                              \
  {                                                                     \
    hipError_t error = status;                                         \
    if (error != hipSuccess) {                                         \
      std::cerr << "Got bad cuda status: " << hipGetErrorString(error) \
                << " at line: " << __LINE__ << std::endl;               \
      exit(EXIT_FAILURE);                                               \
    }                                                                   \
  }


template <unsigned int SHMEM_SIZE_BYTES>
__global__ void increment(float* src, float* dst, unsigned int num_elements)
{
    extern __shared__ float shmem[];
    constexpr unsigned int SHMEM_SIZE = SHMEM_SIZE_BYTES / sizeof(float);
    for (unsigned int i = threadIdx.x; i < num_elements; i += SHMEM_SIZE)
    {
        for (unsigned int j = 0; j < SHMEM_SIZE; j+=blockDim.x)
        {
            shmem[j] = src[i+j];
            shmem[j] += 1;
        }

        for (unsigned int j = 0; j < SHMEM_SIZE; j+=blockDim.x)
        {
            dst[i+j] = src[j];
        }
    }
}

template <unsigned int SHMEM_SIZE_BYTES>
__global__ void increment_async(float* src, float* dst, unsigned int num_elements)
{
    extern __shared__ float shmem[];
    constexpr unsigned int PIPELINE_NUM_STAGES = 2;
    constexpr unsigned int SHMEM_SIZE = SHMEM_SIZE_BYTES / sizeof(float);
    constexpr unsigned int SHMEM_CHUNK_SIZE = SHMEM_SIZE / PIPELINE_NUM_STAGES;
    size_t shmem_offset[2] = {0, SHMEM_CHUNK_SIZE};

    cg::grid_group grid = cg::this_grid();
    cg::thread_block block = cg::this_thread_block();
    
    // threads in the last warp are producers, other threads are consumers
    const cuda::pipeline_role thread_role =
     block.thread_rank() < (block.size() - 32) ? thread_role = cuda::pipeline_role::consumer : cuda::pipeline_role::producer;

    __shared__ cuda::pipeline_shared_state<cuda::thread_scope::thread_scope_block, SHMEM_CHUNK_SIZE> shared_state;
    cuda::pipeline pipeline = cuda::make_pipeline(block, &shared_state);
    
    if (thread_role == cuda::pipeline_role::producer)
    {
        pipeline.prodcer_acquire();
        for (int i = block.thread_rank(); i < SHMEM_CHUNK_SIZE; i += block.size())
        {
            shmem[i] = src[i];
        }
        pipeline.producer_commit();
    }
}









int main()
{
    const unsigned int size = std::pow(2, 24);
    std::vector<float> host_src(size), host_dst(size);
    float* device_src, *device_dst;

    CUDA_CHECK(hipMalloc(&device_src, size * sizeof(float)));
    CUDA_CHECK(hipMalloc(&device_dst, size * sizeof(float)));
    std::fill(host_src.begin(), host_src.end(), 1);
    CUDA_CHECK(hipMemcpy(device_src, host_src.data(), size * sizeof(float), hipMemcpyHostToDevice));

    const unsigned int num_threads = 512;
    const unsigned int num_blocks = 1;
    const unsigned int SHMEM_SIZE = 32768;

    dim3 grid(num_blocks);
    dim3 block(num_threads);
    KernelLogger logger("async");
    increment<SHMEM_SIZE><<<grid, block, SHMEM_SIZE>>>(device_src, device_dst, size);
    for (int i = 0; i < 10; i++)
    {
        logger.Start();
        increment<SHMEM_SIZE><<<grid, block, SHMEM_SIZE>>>(device_src, device_dst, size);
        logger.Stop();
    }



    hipMemcpy(host_dst.data(), device_dst, size * sizeof(float), hipMemcpyDeviceToHost);

    // check result
    for (unsigned int i = 0; i < size; i++)
    {
        if (host_dst[i] != host_src[i] + 1)
        {
            std::cerr << "Mismatch at index: " << i << " expected: " << host_src[i] + 1 << " got: " << host_dst[i] << std::endl;
            exit(EXIT_FAILURE);
        }
    }




}