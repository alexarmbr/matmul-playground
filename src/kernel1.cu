#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <mma.h>

#include "device_utils.cuh"
#include "structs_n_stuff.cuh"


template <unsigned int BM_dim,
unsigned int BN_dim,
unsigned int BK_dim,
unsigned int WM_dim,
unsigned int WN_dim,
unsigned int WK_dim,
unsigned int NUM_THREADS>
__global__ void
kernel_1(half* A,
  half* B,
  half* C,
  half* D,
  const float alpha,
  const float beta,
  const unsigned int M,
  const unsigned int N,
  unsigned int K)
{

  constexpr unsigned int MMA_M_dim = 16;
  constexpr unsigned int MMA_N_dim = 8;
  constexpr unsigned int MMA_K_dim = 8;

  // for convenience/readability in index calculations
  const unsigned int A_stride = K;
  const unsigned int B_stride = N;
  const unsigned int CD_stride = N;

  // loop bounds, constexpr where possible allows for loop unrolling
  constexpr unsigned int mma_tiles_per_warp_k = WK_dim / MMA_K_dim;
  constexpr unsigned int mma_tiles_per_warp_m = WM_dim / MMA_M_dim;
  constexpr unsigned int mma_tiles_per_warp_n = WN_dim / MMA_N_dim;
  constexpr unsigned int warp_tiles_per_block_k = BK_dim / WK_dim;
  const unsigned int num_block_tiles_k = K / BK_dim;
  
  // calculate block/warp indices
  const unsigned int block_m = blockIdx.y;
  const unsigned int block_n = blockIdx.x;
  const unsigned int warp_m = threadIdx.y;
  const unsigned int warp_n = threadIdx.x / 32;
  
  extern __shared__ half shmem[];
  half* A_block_smem = shmem;
  half* B_block_smem = &shmem[BM_dim * BK_dim];

  // declare register storage
  // ptx instructions expect uint32_t registers, where each uint32_t is 2 halfs packed together  
  uint32_t acc_register[mma_tiles_per_warp_m][mma_tiles_per_warp_n][2];
  
  // convenience cast to half for accumulator registers
  half (&acc_register_) [mma_tiles_per_warp_m][mma_tiles_per_warp_n][4] = reinterpret_cast<half(&)[mma_tiles_per_warp_m][mma_tiles_per_warp_n][4]>(acc_register);

  uint32_t A_register[mma_tiles_per_warp_m][mma_tiles_per_warp_k][2];
  uint32_t B_register[mma_tiles_per_warp_k][mma_tiles_per_warp_n];

  // accumulators start at 0
  for (unsigned int mma_m = 0; mma_m < mma_tiles_per_warp_m; mma_m++)
  {
      for (unsigned int mma_n = 0; mma_n < mma_tiles_per_warp_n; mma_n++)
      {
        acc_register_[mma_m][mma_n][0] = 0;
        acc_register_[mma_m][mma_n][1] = 0;
        acc_register_[mma_m][mma_n][2] = 0;
        acc_register_[mma_m][mma_n][3] = 0;
      }
  }

  for (unsigned int block_k = 0; block_k < num_block_tiles_k; block_k++)
  {
    half* A_block_gmem = A + (block_m * BM_dim * A_stride) + (block_k * BK_dim);
    half* B_block_gmem = B + (block_k * BK_dim * B_stride) + (block_n * BN_dim);
    tileMemcpy(A_block_gmem, A_block_smem, K, BM_dim, BK_dim);
    tileMemcpy(B_block_gmem, B_block_smem, N, BK_dim, BN_dim);
    __syncthreads();


    for (unsigned int warp_k = 0; warp_k < warp_tiles_per_block_k; warp_k++)
    {
      
      // preload tiles of a into registers
      half* A_warp_tile = A_block_smem + (warp_m * WM_dim * BK_dim) + (warp_k * WK_dim);
      half* B_warp_tile = B_block_smem + (warp_k * WK_dim * BN_dim) + (warp_n * WN_dim);
      uint32_t A_warp_tile_byte_offset = cvta_to_shared_u32(A_warp_tile);
      uint32_t B_warp_tile_byte_offset = cvta_to_shared_u32(B_warp_tile);

      for (unsigned int mma_m = 0; mma_m < mma_tiles_per_warp_m; mma_m++)
      {
        for (unsigned int mma_k = 0; mma_k < mma_tiles_per_warp_k; mma_k++)
        {
          // byte offset to the top left of the mma tile
          const unsigned int mma_tile_byte_offset = ((mma_m * MMA_M_dim * BK_dim) + (mma_k * MMA_K_dim)) * sizeof(half);
          
          // byte offset to the start of this thread's slice of the mma tile
          const unsigned int thread_byte_offset = (threadIdx.x % MMA_M_dim) * BK_dim * sizeof(half);
          
          // calculate offset in bytes WRT to the start of our shared memory allocation
          const unsigned int thread_offset_bytes = A_warp_tile_byte_offset + mma_tile_byte_offset + thread_byte_offset;
          
          asm volatile (
            "ldmatrix.sync.aligned.m8n8.x2.shared.b16 "
            "{%0, %1}, [%2];"
            : "=r"(A_register[mma_m][mma_k][0]), "=r"(A_register[mma_m][mma_k][1])
            : "r"(thread_offset_bytes)
          );
        }
      }

      // preload tiles of b into registers
      for (unsigned int mma_k = 0; mma_k < mma_tiles_per_warp_k; mma_k++)
      {
        for (unsigned int mma_n = 0; mma_n < mma_tiles_per_warp_n; mma_n++)
        {
          const unsigned int mma_tile_byte_offset = ((mma_k * MMA_K_dim * BN_dim) + (mma_n * MMA_N_dim)) * sizeof(half);
          const unsigned int thread_byte_offset = (threadIdx.x % MMA_K_dim) * BN_dim * sizeof(half);
          const unsigned int thread_offset_bytes = B_warp_tile_byte_offset + mma_tile_byte_offset + thread_byte_offset;
          asm volatile (
            "ldmatrix.sync.aligned.m8n8.x1.trans.shared.b16 "
            "{%0}, [%1];"
            : "=r"(B_register[mma_k][mma_n])
            : "r"(thread_offset_bytes)
        );
        }
      }

      // outer product between mma tiles
      for (unsigned int mma_k = 0; mma_k < mma_tiles_per_warp_k; mma_k++)
      {
        for (unsigned int mma_n = 0; mma_n < mma_tiles_per_warp_n; mma_n++)
        {
          for (unsigned int mma_m = 0; mma_m < mma_tiles_per_warp_m; mma_m++)
          {
            asm volatile (
              "mma.sync.aligned.m16n8k8.row.col.f16.f16.f16.f16 "
              "{%0, %1}, "
              "{%2, %3}, "
              "{%4}, "
              "{%5, %6};"
              : "=r"(acc_register[mma_m][mma_n][0]), "=r"(acc_register[mma_m][mma_n][1])
              : "r"(A_register[mma_m][mma_k][0]), "r"(A_register[mma_m][mma_k][1]),
                "r"(B_register[mma_k][mma_n])
                "r"(acc_register[mma_m][mma_n][0]), "r"(acc_register[mma_m][mma_n][1])
            );
          }
        }
      }
    }
    __syncthreads();
  }

  //////////////
  // epilogue //
  //////////////
  half alpha_ = (half)alpha;
  half beta_ = (half)beta;
  half C_register[mma_tiles_per_warp_m][mma_tiles_per_warp_n][4];
  
  // calculate pointers for this warps C and D tiles
  half* C_block_gmem = C + (block_m * BM_dim * CD_stride) + (block_n * BN_dim);
  half* C_warp_gmem = C_block_gmem + (warp_m * WM_dim * CD_stride) + (warp_n * WN_dim);
  half* D_block_gmem = D + (block_m * BM_dim * CD_stride) + (block_n * BN_dim);
  half* D_warp_gmem = D_block_gmem + (warp_m * WM_dim * CD_stride) + (warp_n * WN_dim);

  for (unsigned int mma_m = 0; mma_m < mma_tiles_per_warp_m; mma_m++)
  {
      for (unsigned int mma_n = 0; mma_n < mma_tiles_per_warp_n; mma_n++)
      {
        half* C_mma_tile = C_warp_gmem + (mma_m * MMA_M_dim * CD_stride) + (mma_n * MMA_N_dim);
        ldmatrix_m16n8_gmem(C_mma_tile, C_register[mma_m][mma_n], N * sizeof(half));
          
        // scale C by beta
        acc_register_[mma_m][mma_n][0] = acc_register_[mma_m][mma_n][0] * alpha_ + C_register[mma_m][mma_n][0] * beta_;
        acc_register_[mma_m][mma_n][1] = acc_register_[mma_m][mma_n][1] * alpha_ + C_register[mma_m][mma_n][1] * beta_;
        acc_register_[mma_m][mma_n][2] = acc_register_[mma_m][mma_n][2] * alpha_ + C_register[mma_m][mma_n][2] * beta_;
        acc_register_[mma_m][mma_n][3] = acc_register_[mma_m][mma_n][3] * alpha_ + C_register[mma_m][mma_n][3] * beta_;
      }
  }

  for (unsigned int mma_m = 0; mma_m < mma_tiles_per_warp_m; mma_m++)
  {
      for (unsigned int mma_n = 0; mma_n < mma_tiles_per_warp_n; mma_n++)
      {
        half* D_mma_tile = D_warp_gmem + (mma_m * MMA_M_dim * CD_stride) + (mma_n * MMA_N_dim);
        stmatrix_m16n8(D_mma_tile, acc_register_[mma_m][mma_n], N * sizeof(half));
      }
  }
}

void kernel_1_launch(sgemm_params device_sgemm_params, KernelLogger& timer, const unsigned int num_runs = 10)
{
    
  constexpr unsigned int BM_dim = 256;
  constexpr unsigned int BN_dim = 128;
  constexpr unsigned int BK_dim = 64;
  
  constexpr unsigned int WARPS_PER_BLOCK_M = 4;
  constexpr unsigned int WARPS_PER_BLOCK_N = 2;
  constexpr unsigned int WARPS_PER_BLOCK_K = 2;

    constexpr unsigned int WM_dim = BM_dim / WARPS_PER_BLOCK_M;
    constexpr unsigned int WN_dim = BN_dim / WARPS_PER_BLOCK_N;
    constexpr unsigned int WK_dim = BK_dim / WARPS_PER_BLOCK_K;

    const unsigned int M = device_sgemm_params.M;
    const unsigned int N = device_sgemm_params.N;
    const unsigned int K = device_sgemm_params.K;

    assert(M % BM_dim == 0);
    assert(N % BN_dim == 0);
    assert(K % BK_dim == 0);
    
    constexpr unsigned int WARP_SIZE = 32;
    const unsigned int BlocksM = M / BM_dim;
    const unsigned int BlocksN = N / BN_dim;
    constexpr unsigned int ThreadsM = WARPS_PER_BLOCK_M;
    constexpr unsigned int ThreadsN = WARP_SIZE * WARPS_PER_BLOCK_N;
    constexpr unsigned int NumThreads = ThreadsM * ThreadsN;
    const unsigned int shmem_bytes = (BM_dim * BK_dim + BK_dim * BN_dim) * sizeof(half);

    dim3 gridDim(BlocksN, BlocksM);
    dim3 blockDim(ThreadsN, ThreadsM);
    
    CUDA_CHECK(hipFuncSetAttribute(reinterpret_cast<const void*>(kernel_1<BM_dim), BN_dim, BK_dim, WM_dim, WN_dim, WK_dim, NumThreads>,
    hipFuncAttributeMaxDynamicSharedMemorySize,
    65536)); // set shared memory limit to 64KB which is maximum for sm_75

    for (int i = 0; i < num_runs; i++)
    {
        timer.Start();
        kernel_1
        <BM_dim, BN_dim, BK_dim,
        WM_dim, WN_dim, WK_dim, NumThreads>
        <<<gridDim, blockDim, shmem_bytes>>>(
            device_sgemm_params.A,
            device_sgemm_params.B,
            device_sgemm_params.C,
            device_sgemm_params.D,
            device_sgemm_params.alpha,
            device_sgemm_params.beta,
            M,
            N,
            K
        );
        timer.Stop();
    }
    double gflops_per_sec = timer.logKernelStats(M, N, K);
    std::cout << gflops_per_sec << " GFLOPS/sec for " << M << "x" << N << "x" << K << std::endl;
    CUDA_CHECK(hipPeekAtLastError());
}


