#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <mma.h>
#include <cute/tensor.hpp>

#include "device_utils.cuh"
#include "structs_n_stuff.cuh"
#include "cute_utils.cuh"

using namespace cute;

template <unsigned int BM_dim,
unsigned int BN_dim,
unsigned int BK_dim,
unsigned int WM_dim,
unsigned int WN_dim,
unsigned int WK_dim,
unsigned int A_swizzle_bits,
unsigned int B_swizzle_bits>
__global__ void
kernel_5(half* A,
  half* B,
  half* C,
  half* D,
  const float alpha,
  const float beta,
  const unsigned int M,
  const unsigned int N,
  unsigned int K)
{

  constexpr unsigned int MMA_M_dim = 16;
  constexpr unsigned int MMA_N_dim = 8;
  constexpr unsigned int MMA_K_dim = 8;

  // loop bounds
  constexpr unsigned int mma_tiles_per_warp_k = WK_dim / MMA_K_dim;
  constexpr unsigned int mma_tiles_per_warp_m = WM_dim / MMA_M_dim;
  constexpr unsigned int mma_tiles_per_warp_n = WN_dim / MMA_N_dim;
  const unsigned int warp_tiles_per_block_k = BK_dim / WK_dim;
  const unsigned int num_block_tiles_k = K / BK_dim;
  
  const unsigned int block_m = blockIdx.y;
  const unsigned int block_n = blockIdx.x;
  const unsigned int warp_m = threadIdx.y;
  const unsigned int warp_n = threadIdx.x / 32;

  auto A_block_tile_shape = make_shape(Int<BM_dim>{}, Int<BK_dim>{});
  auto B_block_tile_shape = make_shape(Int<BK_dim>{}, Int<BN_dim>{});
  auto CD_block_tile_shape = make_shape(Int<BM_dim>{}, Int<BN_dim>{});
  auto A_warp_tile_shape = make_shape(Int<WM_dim>{}, Int<WK_dim>{});
  auto B_warp_tile_shape = make_shape(Int<WK_dim>{}, Int<WN_dim>{});
  auto CD_warp_tile_shape = make_shape(Int<WM_dim>{}, Int<WN_dim>{});
  auto A_mma_tile_shape = make_shape(Int<MMA_M_dim>{}, Int<MMA_K_dim>{});
  auto B_mma_tile_shape = make_shape(Int<MMA_K_dim>{}, Int<MMA_N_dim>{});
  auto CD_mma_tile_shape = make_shape(Int<MMA_M_dim>{}, Int<MMA_N_dim>{});

  

  extern __shared__ half shmem[];
  half* A_smem_ = shmem;
  half* B_smem_ = &shmem[BM_dim * BK_dim];
  // __shared__ half A_smem_[BM_dim * BK_dim];
  // __shared__ half B_smem_[BK_dim * BN_dim];

  Tensor A_gmem = make_tensor(A, make_shape(M, K), LayoutRight{});
  Tensor B_gmem = make_tensor(B, make_shape(K, N), LayoutRight{});
  Tensor C_gmem = make_tensor(C, make_shape(M, N), LayoutRight{});
  Tensor D_gmem = make_tensor(D, make_shape(M, N), LayoutRight{});

  
  auto A_smem_layout = composition(Swizzle<3, 3, A_swizzle_bits>{}, make_layout(A_block_tile_shape, LayoutRight{}));
  auto B_smem_layout = composition(Swizzle<3, 3, B_swizzle_bits>{}, make_layout(B_block_tile_shape, LayoutRight{}));
  Tensor A_smem = make_tensor(make_smem_ptr(A_smem_), A_smem_layout);
  Tensor B_smem = make_tensor(make_smem_ptr(B_smem_), B_smem_layout);

  // block tile each matrix
  Tensor A_block_tiles = zipped_divide(A_gmem, A_block_tile_shape);
  Tensor B_block_tiles = zipped_divide(B_gmem, B_block_tile_shape);
  Tensor C_block_tiles = zipped_divide(C_gmem, CD_block_tile_shape);
  Tensor D_block_tiles = zipped_divide(D_gmem, CD_block_tile_shape);
  
  // create warp tiles for a,b inside of shared memory block tiles
  Tensor A_warp_tiles = zipped_divide(A_smem, A_warp_tile_shape);
  Tensor B_warp_tiles = zipped_divide(B_smem, B_warp_tile_shape);

  // create mma tiles for a,b inside of warp_tiles
  Tensor A_mma_tiles = coalesce(zipped_divide(A_warp_tiles, make_shape(A_mma_tile_shape)), Step<_1,_1>{});
  Tensor B_mma_tiles = coalesce(zipped_divide(B_warp_tiles, make_shape(B_mma_tile_shape)), Step<_1,_1>{});

  // create warp and mma tiles for c,d inside of global memory block tiles
  Tensor C_warp_tiles = coalesce(zipped_divide(C_block_tiles, make_shape(CD_warp_tile_shape)), Step<_1,_1>{});
  Tensor D_warp_tiles = coalesce(zipped_divide(D_block_tiles, make_shape(CD_warp_tile_shape)), Step<_1,_1>{});
  Tensor C_mma_tiles = coalesce(zipped_divide(C_warp_tiles, make_shape(CD_mma_tile_shape)), Step<_1,_1>{});
  Tensor D_mma_tiles = coalesce(zipped_divide(D_warp_tiles, make_shape(CD_mma_tile_shape)), Step<_1,_1>{});
  // C_mma_tiles += 1;

  // declare register storage to hold fragments of C which we will accumulate into
  half C_register[mma_tiles_per_warp_m][mma_tiles_per_warp_n][4];
  if (thread0())
  {
    printf("mma tiles per warp m: %d\n", mma_tiles_per_warp_m);
    printf("mma tiles per warp n: %d\n", mma_tiles_per_warp_n);
    printf("mma tiles per warp k: %d\n", mma_tiles_per_warp_k);
  }



  for (unsigned int mma_m = 0; mma_m < mma_tiles_per_warp_m; mma_m++)
  {
      for (unsigned int mma_n = 0; mma_n < mma_tiles_per_warp_n; mma_n++)
      {
        Tensor C_mma_tile = C_mma_tiles(make_coord(_,_), make_coord(mma_m, mma_n, warp_m, warp_n, block_m, block_n));
        ldmatrix_m16n8_gmem(C_mma_tile.data(), C_register[mma_m][mma_n], N * sizeof(half));
          
          // scale C by beta
          C_register[mma_m][mma_n][0] *= beta;
          C_register[mma_m][mma_n][1] *= beta;
          C_register[mma_m][mma_n][2] *= beta;
          C_register[mma_m][mma_n][3] *= beta;
      }
  }

  for (unsigned int block_k = 0; block_k < num_block_tiles_k; block_k++)
  {
    Tensor A_block_tile = A_block_tiles(make_coord(_,_), make_coord(block_m, block_k));
    Tensor B_block_tile = B_block_tiles(make_coord(_,_), make_coord(block_k, block_n));
    tileMemcpy<BM_dim, BK_dim, half>(A_block_tile.data(), A_smem.data().get(), K, BK_dim);
    tileMemcpy<BK_dim, BN_dim, half>(B_block_tile.data(), B_smem.data().get(), N, BN_dim);

    __syncthreads();

    // preload tiles of a into registers
    for (unsigned int warp_k = 0; warp_k < warp_tiles_per_block_k; warp_k++)
    {
      half A_register[mma_tiles_per_warp_m][mma_tiles_per_warp_k][4];
      for (unsigned int mma_m = 0; mma_m < mma_tiles_per_warp_m; mma_m++)
      {
        for (unsigned int mma_k = 0; mma_k < mma_tiles_per_warp_k; mma_k++)
        {
          Tensor A_mma_tile = A_mma_tiles(make_coord(_,_), make_coord(mma_m, mma_k, warp_m, warp_k));
          ldmatrix_m16n8(A_mma_tile, A_register[mma_m][mma_k]);
        }
      }

      // load one tile of B at a time, and take outer product between this tile and
      // entire warp tile of A
      half B_register[2];
      for (unsigned int mma_k = 0; mma_k < mma_tiles_per_warp_k; mma_k++)
      {
        for (unsigned int mma_n = 0; mma_n < mma_tiles_per_warp_n; mma_n++)
        {
          Tensor B_mma_tile = B_mma_tiles(make_coord(_,_), make_coord(mma_k, mma_n, warp_k, warp_n));
          ldmatrix_n8k8(B_mma_tile, B_register);
          B_register[0] *= alpha;
          B_register[1] *= alpha;
          for (unsigned int mma_m = 0; mma_m < mma_tiles_per_warp_m; mma_m++)
          {
            mma_sync_m16n8k8(
              C_register[mma_m][mma_n],
              A_register[mma_m][mma_k],
              B_register,
              C_register[mma_m][mma_n]
            );
          }
        }
      }
    }
    __syncthreads();
  }

  for (unsigned int mma_m = 0; mma_m < mma_tiles_per_warp_m; mma_m++)
  {
      for (unsigned int mma_n = 0; mma_n < mma_tiles_per_warp_n; mma_n++)
      {
        Tensor D_mma_tile = D_mma_tiles(make_coord(_,_), make_coord(mma_m, mma_n, warp_m, warp_n, block_m, block_n));
        stmatrix_m16n8(D_mma_tile.data(), C_register[mma_m][mma_n], N * sizeof(half));
      }
  }
}

void kernel_5_launch(sgemm_params device_sgemm_params, KernelLogger& timer, const unsigned int num_runs = 10)
{
    
  constexpr unsigned int BM_dim = 128;
  constexpr unsigned int BN_dim = 128;
  constexpr unsigned int BK_dim = 64;
  
  constexpr unsigned int WARPS_PER_BLOCK_M = 4;
  constexpr unsigned int WARPS_PER_BLOCK_N = 4;
  constexpr unsigned int WARPS_PER_BLOCK_K = 2;

    constexpr unsigned int WM_dim = BM_dim / WARPS_PER_BLOCK_M;
    constexpr unsigned int WN_dim = BN_dim / WARPS_PER_BLOCK_N;
    constexpr unsigned int WK_dim = BK_dim / WARPS_PER_BLOCK_K;

    const unsigned int M = device_sgemm_params.M;
    const unsigned int N = device_sgemm_params.N;
    const unsigned int K = device_sgemm_params.K;

    assert(M % BM_dim == 0);
    assert(N % BN_dim == 0);
    assert(K % BK_dim == 0);
    
    constexpr unsigned int WARP_SIZE = 32;
    const unsigned int BlocksM = M / BM_dim;
    const unsigned int BlocksN = N / BN_dim;
    const unsigned int ThreadsM = WARPS_PER_BLOCK_M;
    const unsigned int ThreadsN = WARP_SIZE * WARPS_PER_BLOCK_N;
    const unsigned int shmem_bytes = (BM_dim * BK_dim + BK_dim * BN_dim) * sizeof(half);
    constexpr unsigned int A_swizzle_bits = int_log2(BK_dim/8);
    constexpr unsigned int B_swizzle_bits = int_log2(BN_dim/8);

    dim3 gridDim(BlocksN, BlocksM);
    dim3 blockDim(ThreadsN, ThreadsM);
    
    CUDA_CHECK(hipFuncSetAttribute(reinterpret_cast<const void*>(kernel_5<BM_dim), BN_dim, BK_dim, WM_dim, WN_dim, WK_dim, A_swizzle_bits, B_swizzle_bits>,
    hipFuncAttributeMaxDynamicSharedMemorySize,
    65536)); // set shared memory limit to 64KB which is maximum for sm_75

    for (int i = 0; i < num_runs; i++)
    {
        timer.Start();
        kernel_5
        <BM_dim, BN_dim, BK_dim,
        WM_dim, WN_dim, WK_dim, A_swizzle_bits, B_swizzle_bits>
        <<<gridDim, blockDim, shmem_bytes>>>(
            device_sgemm_params.A,
            device_sgemm_params.B,
            device_sgemm_params.C,
            device_sgemm_params.D,
            device_sgemm_params.alpha,
            device_sgemm_params.beta,
            M,
            N,
            K
        );
        timer.Stop();
    }
    double gflops_per_sec = timer.logKernelStats(M, N, K);
    std::cout << gflops_per_sec << " GFLOPS/sec for " << M << "x" << N << "x" << K << std::endl;
    CUDA_CHECK(hipPeekAtLastError());
}


