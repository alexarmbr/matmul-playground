#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <mma.h>
#include <cute/tensor.hpp>

#include "device_utils.cuh"
#include "structs_n_stuff.cuh"
#include "cute_utils.cuh"

using namespace cute;

template <unsigned int smem_stride>
__device__ __forceinline__ void ldmatrix_b_(
  half* src,
  half (&reg)[8][8][2]
)
{
  uint32_t (&reg_) [8][8] = reinterpret_cast<uint32_t(&)[8][8]>(reg);
  const unsigned int thread_group = (threadIdx.x % 32) / 8;
  const unsigned int thread_row = threadIdx.x % 8;
  const unsigned int thread_offset = (thread_row * smem_stride) + (thread_group * 8);
  const unsigned int swizzled_offset_1 = thread_offset ^ ((thread_offset & 0b1111000000) >> 4);
  const unsigned int swizzled_offset_2 = swizzled_offset_1 ^ 0b111000;
  const uint32_t src_addr_1 = cvta_to_shared_u32(src + swizzled_offset_1);
  const uint32_t src_addr_2 = cvta_to_shared_u32(src + swizzled_offset_2);
  constexpr unsigned int row_offset = smem_stride * 8 * sizeof(half);

  #pragma unroll 8
  for (int block_row = 0; block_row < 8; block_row++)
  {
    asm volatile (
      "ldmatrix.sync.aligned.m8n8.x4.trans.shared.b16 "
      "{%0, %1, %2, %3}, [%4];"
      : "=r"(reg_[block_row][0]), "=r"(reg_[block_row][1]), "=r"(reg_[block_row][2]), "=r"(reg_[block_row][3])
      : "r"(src_addr_1 + block_row * row_offset)
    );
    asm volatile (
      "ldmatrix.sync.aligned.m8n8.x4.trans.shared.b16 "
      "{%0, %1, %2, %3}, [%4];"
      : "=r"(reg_[block_row][7]), "=r"(reg_[block_row][6]), "=r"(reg_[block_row][5]), "=r"(reg_[block_row][4])
      : "r"(src_addr_2 + block_row * row_offset)
    );
  }
}

template <unsigned int smem_stride>
__device__ __forceinline__ void ldmatrix_a_(
  half* src,
  half (&reg)[4][8][4]
)
{
  uint32_t (&reg_) [4][8][2] = reinterpret_cast<uint32_t(&)[4][8][2]>(reg);
  const unsigned int logical_offset_1 = (threadIdx.x % 32) * smem_stride;
  const unsigned int logical_offset_2 = ((threadIdx.x % 32) + 32) * smem_stride;
  const unsigned int swizzled_offset_1 = logical_offset_1 ^ ((logical_offset_1 & 0b111000000) >> 3);
  const unsigned int swizzled_offset_2 = logical_offset_2 ^ ((logical_offset_2 & 0b111000000) >> 3);
  uint32_t src_addr_1 = cvta_to_shared_u32(src + swizzled_offset_1);
  uint32_t src_addr_2 = cvta_to_shared_u32(src + swizzled_offset_2);
  static constexpr int increment_xor_patterns[8] = {
    0b10000,
    0b110000,
    0b10000,
    0b1110000,
    0b10000,
    0b110000,
    0b10000,
    0b1110000
  };

  #pragma unroll 8
  for (int block_col = 0; block_col < 8; block_col++)
  {
      asm volatile (
        "ldmatrix.sync.aligned.m8n8.x4.shared.b16 "
        "{%0, %1, %2, %3}, [%4];"
        : "=r"(reg_[0][block_col][0]), "=r"(reg_[0][block_col][1]), "=r"(reg_[1][block_col][0]), "=r"(reg_[1][block_col][1])
        : "r"(src_addr_1)
    );
      asm volatile (
          "ldmatrix.sync.aligned.m8n8.x4.shared.b16 "
          "{%0, %1, %2, %3}, [%4];"
          : "=r"(reg_[2][block_col][0]), "=r"(reg_[2][block_col][1]), "=r"(reg_[3][block_col][0]), "=r"(reg_[3][block_col][1])
          : "r"(src_addr_2)
    );
    src_addr_1 ^= increment_xor_patterns[block_col];
    src_addr_2 ^= increment_xor_patterns[block_col];
  }
}



template <unsigned int BM_dim,
unsigned int BN_dim,
unsigned int BK_dim,
unsigned int WM_dim,
unsigned int WN_dim,
unsigned int WK_dim,
unsigned int A_swizzle_bits,
unsigned int B_swizzle_bits>
__global__ void
kernel_9(half* A,
  half* B,
  half* C,
  half* D,
  const float alpha,
  const float beta,
  const unsigned int M,
  const unsigned int N,
  unsigned int K)
{

  constexpr unsigned int MMA_M_dim = 16;
  constexpr unsigned int MMA_N_dim = 8;
  constexpr unsigned int MMA_K_dim = 8;

  // loop bounds
  constexpr unsigned int mma_tiles_per_warp_k = WK_dim / MMA_K_dim;
  constexpr unsigned int mma_tiles_per_warp_m = WM_dim / MMA_M_dim;
  constexpr unsigned int mma_tiles_per_warp_n = WN_dim / MMA_N_dim;
  constexpr unsigned int warp_tiles_per_block_k = BK_dim / WK_dim;
  const unsigned int num_block_tiles_k = K / BK_dim;
  
  // const unsigned int blocks_per_M = M / BM_dim;
  const unsigned int blocks_per_N = N / BN_dim;
  // auto swizzle_tile_dim = Int<4>{};
  // const int block_swizzle_tiles_per_M = blocks_per_M / swizzle_tile_dim;
  // const int block_swizzle_tiles_per_N = blocks_per_N / swizzle_tile_dim;
  // Layout block_n_map = make_layout(
  //   make_shape(swizzle_tile_dim, swizzle_tile_dim, block_swizzle_tiles_per_N, block_swizzle_tiles_per_M),
  //   make_stride(1 ,0, swizzle_tile_dim, 0)
  // );

  // Layout block_m_map = make_layout(
  //     make_shape(swizzle_tile_dim, swizzle_tile_dim, block_swizzle_tiles_per_N, block_swizzle_tiles_per_M),
  //     make_stride(0, 1, 0, swizzle_tile_dim)
  // );
  
  // const unsigned int block_m = block_m_map(blockIdx.x);
  // const unsigned int block_n = block_n_map(blockIdx.x);
  const unsigned int block_m = blockIdx.x / blocks_per_N;
  const unsigned int block_n = blockIdx.x % blocks_per_N;
  const unsigned int warp_m = threadIdx.y;
  const unsigned int warp_n = threadIdx.x / 32;

  auto A_block_tile_shape = make_shape(Int<BM_dim>{}, Int<BK_dim>{});
  auto B_block_tile_shape = make_shape(Int<BK_dim>{}, Int<BN_dim>{});
  auto CD_block_tile_shape = make_shape(Int<BM_dim>{}, Int<BN_dim>{});
  auto A_warp_tile_shape = make_shape(Int<WM_dim>{}, Int<WK_dim>{});
  auto B_warp_tile_shape = make_shape(Int<WK_dim>{}, Int<WN_dim>{});
  auto CD_warp_tile_shape = make_shape(Int<WM_dim>{}, Int<WN_dim>{});
  auto A_mma_tile_shape = make_shape(Int<MMA_M_dim>{}, Int<MMA_K_dim>{});
  auto B_mma_tile_shape = make_shape(Int<MMA_K_dim>{}, Int<MMA_N_dim>{});
  auto CD_mma_tile_shape = make_shape(Int<MMA_M_dim>{}, Int<MMA_N_dim>{});

  extern __shared__ half shmem[];
  half* A_smem_ = shmem;
  half* B_smem_ = &shmem[BM_dim * BK_dim];

  Tensor A_gmem = make_tensor(A, make_shape(M, K), LayoutRight{});
  Tensor B_gmem = make_tensor(B, make_shape(K, N), LayoutRight{});
  Tensor C_gmem = make_tensor(C, make_shape(M, N), LayoutRight{});
  Tensor D_gmem = make_tensor(D, make_shape(M, N), LayoutRight{});
  
  // auto A_smem_layout = composition(Swizzle<3, 3, A_swizzle_bits>{}, make_layout(A_block_tile_shape, LayoutRight{}));
  // auto B_smem_layout = composition(Swizzle<3, 3, B_swizzle_bits>{}, make_layout(B_block_tile_shape, LayoutRight{}));
  auto A_smem_layout = make_layout(A_block_tile_shape, LayoutRight{});
  auto B_smem_layout = make_layout(B_block_tile_shape, LayoutRight{});
  Tensor A_smem = make_tensor(make_smem_ptr(A_smem_), A_smem_layout);
  Tensor B_smem = make_tensor(make_smem_ptr(B_smem_), B_smem_layout);

  // block tile each matrix
  Tensor A_block_tiles = zipped_divide(A_gmem, A_block_tile_shape);
  Tensor B_block_tiles = zipped_divide(B_gmem, B_block_tile_shape);
  Tensor C_block_tiles = zipped_divide(C_gmem, CD_block_tile_shape);
  Tensor D_block_tiles = zipped_divide(D_gmem, CD_block_tile_shape);
  
  // create warp tiles for a,b inside of shared memory block tiles
  // Tensor A_warp_tiles = coalesce(zipped_divide(A_smem, A_warp_tile_shape), Step<_1,Step<>>{});
  // Tensor B_warp_tiles = coalesce(zipped_divide(B_smem, B_warp_tile_shape), Step<_1,Step<>>{});
  // Tensor B_warp_tiles = zipped_divide(B_smem, B_warp_tile_shape);
  // if (thread0())
  // {
  //   print(A_warp_tiles.layout());
  // }

  // create mma tiles for a,b inside of warp_tiles
  // Tensor A_mma_tiles = coalesce(zipped_divide(A_warp_tiles, make_shape(A_mma_tile_shape)), Step<_1,Step<>>{});
  // Tensor B_mma_tiles = coalesce(zipped_divide(B_warp_tiles, make_shape(B_mma_tile_shape)), Step<_1,Step<>>{});

  // create warp and mma tiles for c,d inside of global memory block tiles
  Tensor C_warp_tiles = coalesce(zipped_divide(C_block_tiles, make_shape(CD_warp_tile_shape)), Step<_1,_1>{});
  Tensor D_warp_tiles = coalesce(zipped_divide(D_block_tiles, make_shape(CD_warp_tile_shape)), Step<_1,_1>{});
  Tensor C_mma_tiles = coalesce(zipped_divide(C_warp_tiles, make_shape(CD_mma_tile_shape)), Step<_1,_1>{});
  Tensor D_mma_tiles = coalesce(zipped_divide(D_warp_tiles, make_shape(CD_mma_tile_shape)), Step<_1,_1>{});

  // declare register storage for accumulators
  half acc_register[mma_tiles_per_warp_m][mma_tiles_per_warp_n][8];
  for (unsigned int mma_m = 0; mma_m < mma_tiles_per_warp_m; mma_m++)
  {
      for (unsigned int mma_n = 0; mma_n < mma_tiles_per_warp_n; mma_n++)
      {
        acc_register[mma_m][mma_n][0] = 0;
        acc_register[mma_m][mma_n][1] = 0;
        acc_register[mma_m][mma_n][2] = 0;
        acc_register[mma_m][mma_n][3] = 0;
      }
  }

  Tensor A_block_tile = A_block_tiles(make_coord(_,_), make_coord(block_m, 0));
  Tensor B_block_tile = B_block_tiles(make_coord(_,_), make_coord(0, block_n));
  tileMemcpySwizzleUnrolled<BM_dim, BK_dim, A_swizzle_bits>(A_block_tile, A_smem, K, BK_dim);
  tileMemcpySwizzleUnrolled<BK_dim, BN_dim, B_swizzle_bits>(B_block_tile, B_smem, N, BN_dim);
  

  half A_mma_tile_reg[mma_tiles_per_warp_m][mma_tiles_per_warp_k][4];
  half B_mma_tile_reg[mma_tiles_per_warp_k][mma_tiles_per_warp_n][4];
  uint32_t (&A_mma_tile_reg_) [mma_tiles_per_warp_m][mma_tiles_per_warp_k][2] = reinterpret_cast<uint32_t(&)[mma_tiles_per_warp_m][mma_tiles_per_warp_k][2]>(A_mma_tile_reg);
  uint32_t (&B_mma_tile_reg_) [mma_tiles_per_warp_k][mma_tiles_per_warp_n][2] = reinterpret_cast<uint32_t(&)[mma_tiles_per_warp_k][mma_tiles_per_warp_n][2]>(B_mma_tile_reg);
  uint32_t (&acc_register_) [mma_tiles_per_warp_m][mma_tiles_per_warp_n][4] = reinterpret_cast<uint32_t(&)[mma_tiles_per_warp_m][mma_tiles_per_warp_n][4]>(acc_register);

  float4 A_gmem_cache_reg[8];
  float4 B_gmem_cache_reg[4];
  static_assert(BM_dim == 256, "BM_dim must be 256");
  
  // set up pointers into shared memory tile for A
  const half* A_smem_warp_tile_ = A_smem_ + (warp_m * WM_dim) * BK_dim;
  const unsigned int A_logical_offset_1 = (threadIdx.x % 32) * BK_dim;
  const unsigned int A_logical_offset_2 = ((threadIdx.x % 32) + 32) * BK_dim;
  const unsigned int A_swizzled_offset_1 = A_logical_offset_1 ^ ((A_logical_offset_1 & 0b111000000) >> 3);
  const unsigned int A_swizzled_offset_2 = A_logical_offset_2 ^ ((A_logical_offset_2 & 0b111000000) >> 3);
  const uint32_t A_src_addr_1 = cvta_to_shared_u32(A_smem_warp_tile_ + A_swizzled_offset_1);
  const uint32_t A_src_addr_2 = cvta_to_shared_u32(A_smem_warp_tile_ + A_swizzled_offset_2);
  static constexpr int increment_xor_patterns[8] = {
    0b10000,
    0b110000,
    0b10000,
    0b1110000,
    0b10000,
    0b110000,
    0b10000,
    0b1110000
  };

  // set up pointers into shared memory tile for B
  const half* B_smem_warp_tile_ = B_smem_ + (warp_n * WN_dim);
  const unsigned int thread_group = (threadIdx.x % 32) / 8;
  const unsigned int thread_row = threadIdx.x % 8;
  const unsigned int B_logical_offset = (thread_row * BN_dim) + (thread_group * 8);
  const unsigned int B_swizzled_offset_1 = B_logical_offset ^ ((B_logical_offset & 0b1111000000) >> 4);
  const unsigned int B_swizzled_offset_2 = B_swizzled_offset_1 ^ 0b111000;
  const uint32_t B_src_addr_1 = cvta_to_shared_u32(B_smem_warp_tile_ + B_swizzled_offset_1);
  const uint32_t B_src_addr_2 = cvta_to_shared_u32(B_smem_warp_tile_ + B_swizzled_offset_2);
  constexpr unsigned int row_offset = BN_dim * 8 * sizeof(half);

  for (unsigned int block_k = 1; block_k <= num_block_tiles_k; block_k++)
  {
    uint32_t A_src_addr_1_ = A_src_addr_1;
    uint32_t A_src_addr_2_ = A_src_addr_2;
    // #pragma unroll 8
    // for (int block_col = 0; block_col < 8; block_col++)
    // {
      const unsigned int block_col = 0;
        asm volatile (
          "ldmatrix.sync.aligned.m8n8.x4.shared.b16 "
          "{%0, %1, %2, %3}, [%4];"
          : "=r"(A_mma_tile_reg_[0][block_col][0]), "=r"(A_mma_tile_reg_[0][block_col][1]), "=r"(A_mma_tile_reg_[1][block_col][0]), "=r"(A_mma_tile_reg_[1][block_col][1])
          : "r"(A_src_addr_1_)
      );
        asm volatile (
            "ldmatrix.sync.aligned.m8n8.x4.shared.b16 "
            "{%0, %1, %2, %3}, [%4];"
            : "=r"(A_mma_tile_reg_[2][block_col][0]), "=r"(A_mma_tile_reg_[2][block_col][1]), "=r"(A_mma_tile_reg_[3][block_col][0]), "=r"(A_mma_tile_reg_[3][block_col][1])
            : "r"(A_src_addr_2_)
      );
      A_src_addr_1_ ^= increment_xor_patterns[block_col];
      A_src_addr_2_ ^= increment_xor_patterns[block_col];
    // }


    // load first k slice of tiles from b from smem->register
    // #pragma unroll 8
    // for (int block_row = 0; block_row < 8; block_row++)
    // {
      // const unsigned int block_row = 0;
      // asm volatile (
      //   "ldmatrix.sync.aligned.m8n8.x4.trans.shared.b16 "
      //   "{%0, %1, %2, %3}, [%4];"
      //   : "=r"(B_mma_tile_reg_[block_row][0]), "=r"(B_mma_tile_reg_[block_row][1]), "=r"(B_mma_tile_reg_[block_row][2]), "=r"(B_mma_tile_reg_[block_row][3])
      //   : "r"(B_src_addr_1 + block_row * row_offset)
      // );
      // asm volatile (
      //   "ldmatrix.sync.aligned.m8n8.x4.trans.shared.b16 "
      //   "{%0, %1, %2, %3}, [%4];"
      //   : "=r"(B_mma_tile_reg_[block_row][7]), "=r"(B_mma_tile_reg_[block_row][6]), "=r"(B_mma_tile_reg_[block_row][5]), "=r"(B_mma_tile_reg_[block_row][4])
      //   : "r"(B_src_addr_2 + block_row * row_offset)
      // );
    // }


    if (block_k != num_block_tiles_k)
    {
      Tensor A_block_tile = A_block_tiles(make_coord(_,_), make_coord(block_m, block_k));
      Tensor B_block_tile = B_block_tiles(make_coord(_,_), make_coord(block_k, block_n));
      // copy tile of A from global memory to registers
      // we want these memory requests to be in flight while the mmas are being computed
      {
        constexpr unsigned int float4_cols = BK_dim / 8; // 8
        Tensor src_float4 = make_tensor(reinterpret_cast<float4*>(A_block_tile.data()), make_shape(BM_dim, float4_cols), make_stride(K / 8, 1));
        unsigned int thread_idx = threadIdx.y * blockDim.x + threadIdx.x;
        const unsigned int thread_idx_y = thread_idx / float4_cols;
        const unsigned int thread_idx_x = thread_idx % float4_cols;

        A_gmem_cache_reg[0] = src_float4(thread_idx_y, thread_idx_x);
        A_gmem_cache_reg[1] = src_float4(thread_idx_y + 32, thread_idx_x);
        A_gmem_cache_reg[2] = src_float4(thread_idx_y + 64, thread_idx_x);
        A_gmem_cache_reg[3] = src_float4(thread_idx_y + 96, thread_idx_x);
        A_gmem_cache_reg[4] = src_float4(thread_idx_y + 128, thread_idx_x);
        A_gmem_cache_reg[5] = src_float4(thread_idx_y + 160, thread_idx_x);
        A_gmem_cache_reg[6] = src_float4(thread_idx_y + 192, thread_idx_x);
        A_gmem_cache_reg[7] = src_float4(thread_idx_y + 224, thread_idx_x);
      }

      // copy tile of B from global memory to registers
      {
        constexpr unsigned int float4_cols = BN_dim / 8; // 16
        Tensor src_float4 = make_tensor(reinterpret_cast<float4*>(B_block_tile.data()), make_shape(BK_dim, float4_cols), make_stride(N / 8, 1));
        unsigned int thread_idx = threadIdx.y * blockDim.x + threadIdx.x;
        const unsigned int thread_idx_y = thread_idx / float4_cols;
        const unsigned int thread_idx_x = thread_idx % float4_cols;
        B_gmem_cache_reg[0] = src_float4(thread_idx_y, thread_idx_x);
        B_gmem_cache_reg[1] = src_float4(thread_idx_y + 16, thread_idx_x);
        B_gmem_cache_reg[2] = src_float4(thread_idx_y + 32, thread_idx_x);
        B_gmem_cache_reg[3] = src_float4(thread_idx_y + 48, thread_idx_x);
      }
    }
    // ldmatrix_a_<BK_dim>(
    //   A_smem_ + (warp_m * WM_dim) * BK_dim,
    //   A_mma_tile_reg
    // );

    // ldmatrix_b_<BN_dim>(
    //   B_smem_ + (warp_n * WN_dim),
    //   B_mma_tile_reg
    // );


    // outer product between tiles of a and b
    // #pragma unroll
    // for (unsigned int mma_k = 0; mma_k < mma_tiles_per_warp_k; mma_k++)
    // {
    //   #pragma unroll
    //   for (unsigned int mma_n = 0; mma_n < mma_tiles_per_warp_n; mma_n++)
    //   {
    //     #pragma unroll
    //     for (unsigned int mma_m = 0; mma_m < mma_tiles_per_warp_m; mma_m++)
    //     {
    //       // mma_sync_m16n8k8(
    //       //   acc_register[mma_m][mma_n],
    //       //   A_mma_tile_reg[mma_m][mma_k-1],
    //       //   B_mma_tile_reg[mma_k-1][mma_n],
    //       //   acc_register[mma_m][mma_n]
    //       // );
    //         asm volatile (
    //           "mma.sync.aligned.m16n8k8.row.col.f16.f16.f16.f16 "
    //           "{%0, %1}, "
    //           "{%2, %3}, "
    //           "{%4}, "
    //           "{%5, %6};"
    //           : "=r"(acc_register_[mma_m][mma_n][0]), "=r"(acc_register_[mma_m][mma_n][1])
    //           : "r"(A_mma_tile_reg_[mma_m][mma_k][0]), "r"(A_mma_tile_reg_[mma_m][mma_k][1]),
    //             "r"(B_mma_tile_reg_[mma_m][mma_k]),
    //             "r"(acc_register_[mma_m][mma_n][0]), "r"(acc_register_[mma_m][mma_n][1])
    //       );
    //     }
    //   }
    // }

    #pragma unroll
    for (unsigned int mma_k = 0; mma_k < 8; mma_k++)
    {
      #pragma unroll
      for (unsigned int mma_n = 0; mma_n < 4; mma_n++)
      {
        #pragma unroll
        for (unsigned int mma_m = 0; mma_m < 4; mma_m++)
        {
          //   asm volatile (
          //     "mma.sync.aligned.m16n8k8.row.col.f16.f16.f16.f16 "
          //     "{%0, %1}, "
          //     "{%2, %3}, "
          //     "{%4}, "
          //     "{%5, %6};"
          //     : "=r"(acc_register_[mma_m][mma_n][0]), "=r"(acc_register_[mma_m][mma_n][1])
          //     : "r"(A_mma_tile_reg_[mma_m][mma_k][0]), "r"(A_mma_tile_reg_[mma_m][mma_k][1]),
          //       "r"(B_mma_tile_reg_[mma_m][mma_k]),
          //       "r"(acc_register_[mma_m][mma_n][0]), "r"(acc_register_[mma_m][mma_n][1])
          // );
            asm volatile (
              "mma.sync.aligned.m8n8k4.row.col.f16.f16.f16.f16 "
              "{%0, %1, %2, %3}, "
              "{%4, %5}, "
              "{%6, %7}, "
              "{%8, %9, %10, %11};"
              : "=r"(acc_register_[mma_m][mma_n][0]), "=r"(acc_register_[mma_m][mma_n][1]), "=r"(acc_register_[mma_m][mma_n][2]), "=r"(acc_register_[mma_m][mma_n][3])
              : "r"(A_mma_tile_reg_[mma_m][mma_k][0]), "r"(A_mma_tile_reg_[mma_m][mma_k][1]),
                "r"(B_mma_tile_reg_[mma_m][mma_k][0]), "r"(B_mma_tile_reg_[mma_m][mma_k][1]),
                "r"(acc_register_[mma_m][mma_n][0]), "r"(acc_register_[mma_m][mma_n][1]), "r"(acc_register_[mma_m][mma_n][2]), "r"(acc_register_[mma_m][mma_n][3])
            );
        }
      }
    }





    __syncthreads();

    {
      constexpr unsigned int float4_cols = BK_dim / 8; // 8
      auto swizzled_layout = composition(Swizzle<3,0,A_swizzle_bits>{}, make_layout(make_shape(BM_dim, float4_cols), make_stride(BK_dim / 8, 1)));
      // auto dst_layout = make_layout(make_shape(BM_dim, float4_cols), make_stride(BK_dim / 8, 1));
      Tensor dst_float4 = make_tensor(reinterpret_cast<float4*>(A_smem.data().get()), swizzled_layout);
      unsigned int thread_idx = threadIdx.y * blockDim.x + threadIdx.x;
      unsigned int thread_idx_y = thread_idx / float4_cols;
      unsigned int thread_idx_x = thread_idx % float4_cols;
      dst_float4(thread_idx_y, thread_idx_x) = A_gmem_cache_reg[0];
      dst_float4(thread_idx_y + 32, thread_idx_x) = A_gmem_cache_reg[1];
      dst_float4(thread_idx_y + 64, thread_idx_x) = A_gmem_cache_reg[2];
      dst_float4(thread_idx_y + 96, thread_idx_x) = A_gmem_cache_reg[3];
      dst_float4(thread_idx_y + 128, thread_idx_x) = A_gmem_cache_reg[4];
      dst_float4(thread_idx_y + 160, thread_idx_x) = A_gmem_cache_reg[5];
      dst_float4(thread_idx_y + 192, thread_idx_x) = A_gmem_cache_reg[6];
      dst_float4(thread_idx_y + 224, thread_idx_x) = A_gmem_cache_reg[7];
    }

    {
      constexpr unsigned int float4_cols = BN_dim / 8; // 16
      auto swizzled_layout = composition(Swizzle<3,0,B_swizzle_bits>{}, make_layout(make_shape(BK_dim, float4_cols), make_stride(BN_dim / 8, 1)));
      // auto dst_layout = make_layout(make_shape(BK_dim, float4_cols), make_stride(BN_dim / 8, 1));
      unsigned int thread_idx = threadIdx.y * blockDim.x + threadIdx.x;
      unsigned int thread_idx_y = thread_idx / float4_cols;
      unsigned int thread_idx_x = thread_idx % float4_cols;
      Tensor dst_float4 = make_tensor(reinterpret_cast<float4*>(B_smem.data().get()), swizzled_layout);
      dst_float4(thread_idx_y, thread_idx_x) = B_gmem_cache_reg[0];
      dst_float4(thread_idx_y + 16, thread_idx_x) = B_gmem_cache_reg[1];
      dst_float4(thread_idx_y + 32, thread_idx_x) = B_gmem_cache_reg[2];
      dst_float4(thread_idx_y + 48, thread_idx_x) = B_gmem_cache_reg[3];
    }
  }

  half alpha_ = (half)alpha;
  half beta_ = (half)beta;
  half C_register[mma_tiles_per_warp_m][mma_tiles_per_warp_n][4];
  for (unsigned int mma_m = 0; mma_m < mma_tiles_per_warp_m; mma_m++)
  {
      for (unsigned int mma_n = 0; mma_n < mma_tiles_per_warp_n; mma_n++)
      {
        Tensor C_mma_tile = C_mma_tiles(make_coord(_,_), make_coord(mma_m, mma_n, warp_m, warp_n, block_m, block_n));
        ldmatrix_m16n8_gmem(C_mma_tile.data(), C_register[mma_m][mma_n], N * sizeof(half));
        acc_register[mma_m][mma_n][0] = acc_register[mma_m][mma_n][0] * alpha_ + C_register[mma_m][mma_n][0] * beta_;
        acc_register[mma_m][mma_n][1] = acc_register[mma_m][mma_n][1] * alpha_ + C_register[mma_m][mma_n][1] * beta_;
        acc_register[mma_m][mma_n][2] = acc_register[mma_m][mma_n][2] * alpha_ + C_register[mma_m][mma_n][2] * beta_;
        acc_register[mma_m][mma_n][3] = acc_register[mma_m][mma_n][3] * alpha_ + C_register[mma_m][mma_n][3] * beta_;
      }
  }

  // for (unsigned int mma_m = 0; mma_m < mma_tiles_per_warp_m; mma_m++)
  // {
  //     for (unsigned int mma_n = 0; mma_n < mma_tiles_per_warp_n; mma_n++)
  //     {
  //       Tensor D_mma_tile = D_mma_tiles(make_coord(_,_), make_coord(mma_m, mma_n, warp_m, warp_n, block_m, block_n));
  //       stmatrix_m16n8(D_mma_tile.data(), acc_register[mma_m][mma_n], N * sizeof(half));
  //     }
  // }
}

void kernel_9_launch(sgemm_params device_sgemm_params, KernelLogger& timer, const unsigned int num_runs = 10)
{
    
  constexpr unsigned int BM_dim = 256;
  constexpr unsigned int BN_dim = 128;
  constexpr unsigned int BK_dim = 64;
  
  constexpr unsigned int WARPS_PER_BLOCK_M = 4;
  constexpr unsigned int WARPS_PER_BLOCK_N = 2;
  constexpr unsigned int WARPS_PER_BLOCK_K = 1;

    constexpr unsigned int WM_dim = BM_dim / WARPS_PER_BLOCK_M;
    constexpr unsigned int WN_dim = BN_dim / WARPS_PER_BLOCK_N;
    constexpr unsigned int WK_dim = BK_dim / WARPS_PER_BLOCK_K;

    const unsigned int M = device_sgemm_params.M;
    const unsigned int N = device_sgemm_params.N;
    const unsigned int K = device_sgemm_params.K;

    assert(M % BM_dim == 0);
    assert(N % BN_dim == 0);
    assert(K % BK_dim == 0);
    
    constexpr unsigned int WARP_SIZE = 32;
    const unsigned int BlocksM = M / BM_dim;
    const unsigned int BlocksN = N / BN_dim;
    const unsigned int ThreadsM = WARPS_PER_BLOCK_M;
    const unsigned int ThreadsN = WARP_SIZE * WARPS_PER_BLOCK_N;
    const unsigned int shmem_bytes = (BM_dim * BK_dim + BK_dim * BN_dim) * sizeof(half);
    constexpr unsigned int A_swizzle_bits = int_log2(BK_dim/8);
    constexpr unsigned int B_swizzle_bits = int_log2(BN_dim/8);

    dim3 gridDim(BlocksN * BlocksM, 1);
    dim3 blockDim(ThreadsN, ThreadsM);
    
    CUDA_CHECK(hipFuncSetAttribute(reinterpret_cast<const void*>(kernel_9<BM_dim), BN_dim, BK_dim, WM_dim, WN_dim, WK_dim, A_swizzle_bits, B_swizzle_bits>,
    hipFuncAttributeMaxDynamicSharedMemorySize,
    65536)); // set shared memory limit to 64KB which is maximum for sm_75

    for (int i = 0; i < num_runs; i++)
    {
        timer.Start();
        kernel_9
        <BM_dim, BN_dim, BK_dim,
        WM_dim, WN_dim, WK_dim, A_swizzle_bits, B_swizzle_bits>
        <<<gridDim, blockDim, shmem_bytes>>>(
            device_sgemm_params.A,
            device_sgemm_params.B,
            device_sgemm_params.C,
            device_sgemm_params.D,
            device_sgemm_params.alpha,
            device_sgemm_params.beta,
            M,
            N,
            K
        );
        timer.Stop();
    }
    double gflops_per_sec = timer.logKernelStats(M, N, K);
    std::cout << gflops_per_sec << " GFLOPS/sec for " << M << "x" << N << "x" << K << std::endl;
    CUDA_CHECK(hipPeekAtLastError());
}


