#include <hipblas.h>

#include "structs_n_stuff.cuh"

void cublas_launch(sgemm_params device_sgemm_params, KernelLogger& timer, const unsigned int num_runs = 10)
{
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    const half alpha = device_sgemm_params.alpha;
    const half beta = device_sgemm_params.beta;
    const int M = device_sgemm_params.M;
    const int N = device_sgemm_params.N;
    const int K = device_sgemm_params.K;

    // warmup
    hipblasStatus_t status = hipblasHgemm(handle,
        HIPBLAS_OP_N,
        HIPBLAS_OP_N,
        M,
        N,
        K,
        &alpha,
        device_sgemm_params.A,
        K,
        device_sgemm_params.B,
        N,
        &beta,
        device_sgemm_params.C,
        N
    );


    for (int i = 0; i < num_runs; i++)
    {
        timer.Start();
        hipblasStatus_t status = hipblasHgemm(handle,
            HIPBLAS_OP_N,
            HIPBLAS_OP_N,
            M,
            N,
            K,
            &alpha,
            device_sgemm_params.A,
            K,
            device_sgemm_params.B,
            N,
            &beta,
            device_sgemm_params.C,
            N
        );
        timer.Stop();

        if (status != HIPBLAS_STATUS_SUCCESS)
        {
            throw std::runtime_error("cuBLAS kernel failed");
        }
    }
    double gflops_per_sec = timer.logKernelStats(M, N, K);
    std::cout << "cuBLAS: " << gflops_per_sec << " GFLOPS/sec for " << M << "x" << N << "x" << K << std::endl;
}

